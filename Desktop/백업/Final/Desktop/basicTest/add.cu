__global__ void add(int *a, int *b, int *c) {
	*c= *a + *b;
}


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

int main(void) {
	int a, b, c;
	int *dev_a, *dev_b, *dev_c;
	int size = sizeof(int);

	//allocate device copies of a, b, c
	hipMalloc((void **)&dev_a, size);
	hipMalloc((void **)&dev_b, size);
	hipMalloc((void **)&dev_c, size);

	a=2;
	b=7;

	//copy inputs to device
	hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(dev_a, dev_b, dev_c);

	//copy device result back to host copy of c
	hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	printf("the value of c is %d\n", c);
	return 0;
}
